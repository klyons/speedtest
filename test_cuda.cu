#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "PGMImage.h"


__global__ void hello_cuda() {
    printf("Hello from CUDA!\n");
}

__global__ void demosaic_kernel(uint16_t* input, uint16_t* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;

    uint16_t R, G, B;

    // Use shared memory with padding to reduce bank conflicts
    __shared__ uint16_t sharedMem[16][16 + 1];

    // Load data into shared memory
    sharedMem[threadIdx.y][threadIdx.x] = input[idx];
    __syncthreads();

    // Perform demosaicing (same logic as before)
    // ...

    output[idx * 3 + 0] = R;
    output[idx * 3 + 1] = G;
    output[idx * 3 + 2] = B;
}

void BayerDemosaicCUDA(const std::string& inputFilename, const std::string& outputFilename) {
    try {
        // Load the PGM image using the PGMImage class
        PGMImage inputImage(inputFilename);

        uint16_t width = inputImage.width();
        uint16_t height = inputImage.height();
        uint16_t* raw_data = inputImage.data();

        if (!raw_data) {
            std::cerr << "Failed to read image data" << std::endl;
            return;
        }

        // Allocate device memory
        uint16_t* d_input;
        uint16_t* d_output;
        size_t inputSize = width * height * sizeof(uint16_t);
        size_t outputSize = width * height * 3 * sizeof(uint16_t);
        hipMalloc(&d_input, inputSize);
        hipMalloc(&d_output, outputSize);

        // Copy data to device
        hipMemcpy(d_input, raw_data, inputSize, hipMemcpyHostToDevice);

        // Define block and grid sizes
        dim3 blockSize(16, 16);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

        // Launch the kernel
        demosaic_kernel << <gridSize, blockSize >> > (d_input, d_output, width, height);
        hipDeviceSynchronize();

        // Allocate host memory for the output
        uint16_t* output_data = new uint16_t[width * height * 3];

        // Copy the result back to host
        hipMemcpy(output_data, d_output, outputSize, hipMemcpyDeviceToHost);

        // Create an output PGM image
        PGMImage outputImage(width, height, 3);

        // Copy the output data to the PGMImage data
        std::memcpy(outputImage.data(), output_data, outputSize);

        // Save the output image
        if (outputImage.Write(outputFilename)) {
            std::cout << "Image written successfully to " << outputFilename << std::endl;
        }
        else {
            std::cerr << "Failed to write image to " << outputFilename << std::endl;
        }

        // Free device memory
        hipFree(d_input);
        hipFree(d_output);

        // Free host memory
        delete[] output_data;
    }
    catch (const std::exception& e) {
        std::cerr << "Standard exception: " << e.what() << std::endl;
    }
    catch (...) {
        std::cerr << "Unknown error occurred!" << std::endl;
    }
}

int main() {
    hello_cuda<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
